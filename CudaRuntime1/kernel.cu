#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_texture_types.h"
#include ""
#include "ImageFloatConverter.h"
#include "KernelGenerator.h"
#include "ReadWriteImg.h"
#include <algorithm>
#include <stdio.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <texture_types.h>

struct DimParams
{
    dim3 gridDim, blockDim;
    unsigned sharedMemSize = 0;
};

template <class KerType, class... KerArgs>
float testKernel(KerType ker, const DimParams& dimParams, KerArgs&&... _Args)
{
    auto gridSize = dimParams.gridDim;
    auto blockSize = dimParams.blockDim;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // запись события
    hipEventRecord(start, 0);
    if (dimParams.sharedMemSize == 0)
        ker << <gridSize, blockSize >> > (std::forward<KerArgs>(_Args)...);
    else
        ker << <gridSize, blockSize, dimParams.sharedMemSize >> > (std::forward<KerArgs>(_Args)...);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

std::pair<Image<byte>::Images, Image<byte>::Images> splitImages(const Image<byte>& src,
    const Image<byte>& dest, int devCount);

DimParams getDimParams(int devIdx, const Image<byte>& src);

template <class KerType>
float testRegularKernelMultipleDevs(KerType ker, const Image<byte>& source, Image<byte>& res);

template <class KerType>
float testTextureKernelMultipleDevs(KerType ker, const Image<byte>& source, Image<byte>& res);

hipTextureObject_t create3DTextureObject(const Image<byte>& img);


__global__ 
void conv(byte* img, unsigned imgWidth, unsigned imgHeight, unsigned imgCh, float* kernel, int kernelSize, byte* resImg)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int ch = blockIdx.z;
    if (ch != 3)
    {
        float sum = 0.0;

        for (int cx = -kernelSize / 2; cx < kernelSize / 2 + 1; ++cx) {
            for (int cy = -kernelSize / 2; cy < kernelSize / 2 + 1; ++cy) {
                if (x + cx >= 0 && x + cx < imgWidth && y + cy >= 0 && y + cy < imgHeight) {
                    const int imgIdx = ((y + cy) * imgWidth + x + cx) * imgCh + ch;
                    const int kernelIdx = (cy + kernelSize / 2) * kernelSize + cx + kernelSize / 2;
                    sum += kernel[kernelIdx] * img[imgIdx];
                }
            }
        }
        resImg[(y * imgWidth + x) * imgCh + ch] = sum;
    }
    else
    {
        const int idx = (y * imgWidth + x) * imgCh + ch;
        resImg[idx] = img[idx];
    }
}

__global__
void convTex(hipTextureObject_t tex, unsigned imgWidth,
    unsigned imgHeight, unsigned imgCh, float* kernel, int kernelSize, byte* resImg)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int ch = blockIdx.z;
    if (ch != 3)
    {
        float sum = 0.0;

        for (int cx = -kernelSize / 2; cx < kernelSize / 2 + 1; ++cx) {
            for (int cy = -kernelSize / 2; cy < kernelSize / 2 + 1; ++cy) {
                /*if (x + cx >= 0 && x + cx < imgWidth && y + cy >= 0 && y + cy < imgHeight)*/ {
                    byte imgVal = tex3D<byte>(tex, x + cx, y + cy, ch);
                    const int kernelIdx = (cy + kernelSize / 2) * kernelSize + cx + kernelSize / 2;
                    sum += kernel[kernelIdx] * imgVal;
                }
            }
        }
        resImg[(y * imgWidth + x) * imgCh + ch] = sum;
    }
    else
    {
        const int idx = (y * imgWidth + x) * imgCh + ch;
        resImg[idx] = tex3D<byte>(tex, x, y, ch);
    }
}

__global__
void convShKernel(byte* img, unsigned imgWidth, unsigned imgHeight, unsigned imgCh, float* kernel, int kernelSize, byte* resImg)
{
    __shared__ float kernelSh[32 * 32];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int ch = blockIdx.z;

    if (threadIdx.x < kernelSize && threadIdx.y < kernelSize && ch == 0)
    {
        const int idx = threadIdx.y * kernelSize + threadIdx.x;
        kernelSh[idx] = kernel[idx];
    }
    __syncthreads();
    if (ch != 3)
    {
        float sum = 0.0;

        for (int cx = -kernelSize / 2; cx < kernelSize / 2 + 1; ++cx) {
            for (int cy = -kernelSize / 2; cy < kernelSize / 2 + 1; ++cy) {
                if (x + cx >= 0 && x + cx < imgWidth && y + cy >= 0 && y + cy < imgHeight) {
                    int imgIdx = ((y + cy) * imgWidth + x + cx) * imgCh + ch;
                    int kernelIdx = (cy + kernelSize / 2) * kernelSize + cx + kernelSize / 2;
                    sum += kernelSh[kernelIdx] * img[imgIdx];
                }
            }
        }
        resImg[(y * imgWidth + x) * imgCh + ch] = sum;
    }
    else
    {
        int idx = (y * imgWidth + x) * imgCh + ch;
        resImg[idx] = img[idx];
    }
}

__global__ 
void convSh(byte* img, unsigned imgWidth, unsigned imgHeight, unsigned imgCh, float* kernel, int kernelSize, byte* resImg)
{
    const int kernelRadius = (kernelSize - 1) / 2;
    const int x0 = blockIdx.x * blockDim.x;
    const int y0 = blockIdx.y * blockDim.y;
    const int x = x0 + threadIdx.x;
    const int y = y0 + threadIdx.y;
    const int ch = blockIdx.z;
    const int tileWidth = blockDim.x + kernelSize - 1;
    const int tileHeight = blockDim.y + kernelSize - 1;
    const int tileSize = tileWidth * tileHeight;

    extern __shared__ byte blockTile[];
    const int blockSize = blockDim.x * blockDim.y;
    const int count = tileSize / blockSize;
    const int threadNum = threadIdx.y * blockDim.x + threadIdx.x;
    int curTileIdx;
    for (int i = 0; i < count; ++i)
    {
        curTileIdx = threadNum * count + i;
        const int imgX = x0 - kernelRadius + curTileIdx % tileWidth;
        const int imgY = y0 - kernelRadius + curTileIdx / tileWidth;
        if (imgX < 0 || imgX >= imgWidth || imgY < 0 || imgY >= imgHeight)
            blockTile[curTileIdx] = 0;
        else
            blockTile[curTileIdx] = img[(imgY * imgWidth + imgX) * imgCh + ch];
    }
    curTileIdx = count * blockSize + threadNum;
    if (tileSize > curTileIdx)
    {
        const int imgX = x0 - kernelRadius + curTileIdx % tileWidth;
        const int imgY = y0 - kernelRadius + curTileIdx / tileWidth;
        if (imgX < 0 || imgX >= imgWidth || imgY < 0 || imgY >= imgHeight)
            blockTile[count * blockSize + threadNum] = 0;
        else
            blockTile[count * blockSize + threadNum] = img[(imgY * imgWidth + imgX) * imgCh + ch];
    }
    const int invId = blockSize - threadNum - 1;
    float* kernelSh = (float*)(blockTile + tileSize * sizeof(byte));
    if (invId < kernelSize * kernelSize)
    {
        kernelSh[invId] = kernel[invId];
    }
    __syncthreads();
    if (ch != 3)
    {
        float sum = 0.0;

        for (int cx = -kernelRadius; cx < kernelRadius + 1; ++cx) {
            for (int cy = -kernelRadius; cy < kernelRadius + 1; ++cy) {
                {
                    int tileIdx = (kernelRadius + cy + threadIdx.y) * tileWidth +
                        cx + threadIdx.x + kernelRadius;
                    int kernelIdx = (cy + kernelRadius) * kernelSize + cx + kernelRadius;
                    sum += kernelSh[kernelIdx] * blockTile[tileIdx];
                }
            }
        }
        resImg[(y * imgWidth + x) * imgCh + ch] = sum;
    }
    else
    {
        int idx = (y * imgWidth + x) * imgCh + ch;
        int tileIdx = (kernelRadius + threadIdx.y) * tileWidth +
            threadIdx.x + kernelRadius;
        resImg[idx] = blockTile[tileIdx];
    }
}



int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    ReadWriteImg reader;
    const std::string base = "C:\\Users\\User\\Documents\\Study\\GPU\\ImgProc";
    const std::string img_input_name = "test.png";
    const std::string img_output_name = "res.png";
    auto _img = reader.readImage(base + '\\' + img_input_name);
    
    auto _resImg = _img.createSimilar();
    auto images = _img.splitImage(deviceCount);
    auto resImages = _resImg.splitImage(deviceCount);
    Kernel ker = KernelGenerator().generateBlurKernel(17);

    for (int i = 0; i < deviceCount; ++i)
    {
        auto& img = images[i];
        auto& resImg = resImages[i];
        hipSetDevice(i);

        byte *imgD, *resD;
        float* kernelD;
        hipMalloc(&imgD, img.getSize());
        hipMemcpy(imgD, img.data, img.getSize(), hipMemcpyHostToDevice);

        hipMalloc(&resD, img.getSize());

        unsigned size2 = ker.size * ker.size * sizeof(float);
        hipMalloc(&kernelD, size2);
        auto res = hipMemcpy(kernelD, ker.data, size2, hipMemcpyHostToDevice);

        unsigned numOfThreadsPerDim = 32;

        hipTextureObject_t texObj = create3DTextureObject(img);


        dim3 blockDim, gridDim;
        blockDim.x = std::min(numOfThreadsPerDim, img.width);
        blockDim.y = std::min(numOfThreadsPerDim, img.height);
        gridDim.x = (img.width + blockDim.x - 1) / blockDim.x;
        gridDim.y = (img.height + blockDim.y - 1) / blockDim.y;
        gridDim.z = img.channels;

        unsigned memSize = (numOfThreadsPerDim + ker.size - 1) * 
            (numOfThreadsPerDim + ker.size - 1) * sizeof(byte) +
            size2;

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        if (memSize > deviceProp.sharedMemPerBlock)
        {
            printf("Not enough shared memory\n");
            return 1;
        }

        DimParams dimParams;
        dimParams.blockDim = blockDim;
        dimParams.gridDim = gridDim;

        //test common memory convolution
        float elapsedTimeCommonMemory = testKernel(conv, dimParams, imgD, img.width,
            img.height, img.channels, kernelD, ker.size, resD);
        printf("Elapsed time for common memory: %f\n", elapsedTimeCommonMemory);

        //test texture memory convolution
        elapsedTimeCommonMemory = testKernel(convTex, dimParams, texObj, img.width,
            img.height, img.channels, kernelD, ker.size, resD);
        printf("Elapsed time for texture memory: %f\n", elapsedTimeCommonMemory);

        //test shared memory convolution
        elapsedTimeCommonMemory = testKernel(convShKernel, dimParams, imgD, img.width,
            img.height, img.channels, kernelD, ker.size, resD);
        printf("Elapsed time for shared kernel: %f\n", elapsedTimeCommonMemory);

        //test shared memory convolution
        dimParams.sharedMemSize = memSize;
        elapsedTimeCommonMemory = testKernel(convSh, dimParams, imgD, img.width,
            img.height, img.channels, kernelD, ker.size, resD);
        printf("Elapsed time for shared memory: %f\n", elapsedTimeCommonMemory);


        hipMemcpy(resImg.data, resD, resImg.getSize(), hipMemcpyDeviceToHost);

        hipFree(imgD);
        hipFree(resD);
        hipFree(kernelD);
        hipDestroyTextureObject(texObj);
    }
    //auto resImg = _img;
    reader.writeImage(_resImg, base + '\\' + img_output_name);
    _resImg.clear();
    _img.clear();
    free(ker.data);
    return 0;
}

std::pair<Image<byte>::Images, Image<byte>::Images> splitImages(const Image<byte>& src, const Image<byte>& dest, int devCount)
{
    return std::pair<Image<byte>::Images, Image<byte>::Images>();
}

hipTextureObject_t create3DTextureObject(const Image<byte>& img) {

    // Размеры 3D текстуры (ширина, высота, число каналов)
    const int width = img.width,
        height = img.height,
        channels = img.channels;
    hipExtent extent = make_hipExtent(width, height, channels);

    // Описание формата канала
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<byte>();

    // Создание 3D массива
    hipArray_t cuArray3D;
    hipMalloc3DArray(&cuArray3D, &channelDesc, extent);

    // Преобразование данных в формат, совместимый с 3D массивом
    std::vector<byte> linearData(img.getSize());
    for (int z = 0; z < channels; ++z) {
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                int idx = (y * width + x) * channels + z;
                int linearIdx = z * (width * height) + y * width + x;
                linearData[linearIdx] = img.data[idx];
            }
        }
    }

    // Копирование данных в 3D массив
    hipMemcpy3DParms copyParams = {};
    copyParams.srcPtr = make_hipPitchedPtr(
        linearData.data(), width * sizeof(unsigned char), width, height);
    copyParams.dstArray = cuArray3D;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    // Настройка ресурса текстуры
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray3D;

    // Настройка параметров текстуры
    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeClamp; // Адресация по ширине
    texDesc.addressMode[1] = hipAddressModeClamp; // Адресация по высоте
    texDesc.addressMode[2] = hipAddressModeClamp; // Адресация по глубине (каналы)
    texDesc.filterMode = hipFilterModePoint;     // Без интерполяции
    texDesc.readMode = hipReadModeElementType;    // Чтение байтов без нормализации
    texDesc.normalizedCoords = false;             // Не нормализовать координаты

    // Создание текстурного объекта
    hipTextureObject_t texObject = 0;
    hipCreateTextureObject(&texObject, &resDesc, &texDesc, nullptr);

    return texObject;
}

/*#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include "ImageFloatConverter.h"
#include "KernelGenerator.h"
#include "ReadWriteImg.h"

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s (%s)\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Измерение времени выполнения ядра
float measureKernelExecution(
    const dim3& gridDim, const dim3& blockDim,
    void (*kernel)(...), void* args,
    const char* kernelName) {

    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate(start)");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate(stop)");

    checkCudaError(hipEventRecord(start, 0), "hipEventRecord(start)");

    // Запуск ядра
    kernel << <gridDim, blockDim >> > (args);
    checkCudaError(hipGetLastError(), kernelName);

    // Синхронизация
    checkCudaError(hipEventRecord(stop, 0), "hipEventRecord(stop)");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize");

    float elapsedTime = 0.0f;
    checkCudaError(hipEventElapsedTime(&elapsedTime, start, stop), "hipEventElapsedTime");

    printf("Kernel '%s' execution time: %.2f ms\n", kernelName, elapsedTime);

    // Освобождение ресурсов
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}

int main() {
    int deviceCount;
    checkCudaError(hipGetDeviceCount(&deviceCount), "hipGetDeviceCount");

    ReadWriteImg reader;
    std::string base = "C:\\\\Users\\\\User\\\\Documents\\\\Study\\\\GPU\\\\ImgProc";
    std::string img1_name = "test.png";
    std::string img2_name = "res.png";

    auto _img = reader.readImage(base + "\\" + img1_name);
    auto images = _img.splitImage(deviceCount);
    auto _resImg = _img.createSimilar();
    auto resImages = _resImg.splitImage(deviceCount);

    Kernel ker = KernelGenerator().generateBlurKernel(75);

    for (int i = 0; i < deviceCount; ++i) {
        hipSetDevice(i);

        auto& img = images[i];
        auto& resImg = resImages[i];

        byte* imgD, * resD;
        float* kernelD;

        checkCudaError(hipMalloc(&imgD, img.getSize()), "hipMalloc(imgD)");
        checkCudaError(hipMemcpy(imgD, img.data, img.getSize(), hipMemcpyHostToDevice), "hipMemcpy(imgD)");

        checkCudaError(hipMalloc(&resD, img.getSize()), "hipMalloc(resD)");
        checkCudaError(hipMalloc(&kernelD, ker.size * ker.size * sizeof(float)), "hipMalloc(kernelD)");
        checkCudaError(hipMemcpy(kernelD, ker.data, ker.size * ker.size * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy(kernelD)");

        dim3 blockDim(32, 32);
        dim3 gridDim(
            (img.width + blockDim.x - 1) / blockDim.x,
            (img.height + blockDim.y - 1) / blockDim.y,
            img.channels
        );

        // Пример вызова замера времени
        measureKernelExecution(gridDim, blockDim, conv, { imgD, ... }, "conv");

        checkCudaError(hipMemcpy(resImg.data, resD, resImg.getSize(), hipMemcpyDeviceToHost), "hipMemcpy(resD)");

        hipFree(imgD);
        hipFree(resD);
        hipFree(kernelD);
    }

    reader.writeImage(_resImg, base + "\\" + img2_name);
    _img.clear();
    _resImg.clear();
    return 0;
}
*/