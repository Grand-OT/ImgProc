#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_texture_types.h"
#include ""
#include "ImageFloatConverter.h"
#include "KernelGenerator.h"
#include "ReadWriteImg.h"
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <map>
#include <mutex>
#include <stdio.h>
#include <>
#include <texture_types.h>
#include <thread>

#define INITIALIZE_ENTITIES(i)  auto& img = images[i]; \
                                auto& resImg = resImages[i]; \
                                auto& imgD = imgDs[i]; \
                                auto& resD = resDs[i]; \
                                auto& kernelD = kernelDs[i]; \
                                auto& blockDim = blockDims[i]; \
                                auto& gridDim = gridDims[i]; \
                                auto& memSize = memSizes[i]; \
                                auto& dimParams = dimParamsS[i]; \
                                auto& texObj = texObjs[i];

struct DimParams
{
    dim3 gridDim, blockDim;
    unsigned sharedMemSize = 0;
};

struct Measurements
{
    using MeasRes = std::map<int, double>;
    std::mutex mtx;
    MeasRes meas[4];
};

Measurements results[2];

void writeToFile(const std::string& filename);

template <class KerType, class... KerArgs>
float testKernel(KerType ker, const DimParams& dimParams, KerArgs&&... _Args)
{
    auto gridSize = dimParams.gridDim;
    auto blockSize = dimParams.blockDim;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // запись события
    hipEventRecord(start, 0);
    if (dimParams.sharedMemSize == 0)
        ker << <gridSize, blockSize >> > (std::forward<KerArgs>(_Args)...);
    else
        ker << <gridSize, blockSize, dimParams.sharedMemSize >> > (std::forward<KerArgs>(_Args)...);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

std::pair<Image<byte>::Images, Image<byte>::Images> splitImages(const Image<byte>& src,
    const Image<byte>& dest, int devCount);

hipTextureObject_t create3DTextureObject(const Image<byte>& img);


__global__ 
void conv(const Image<byte> source, Image<byte> res, const Kernel kernel)
{
    
    const int resx = blockIdx.x * blockDim.x + threadIdx.x;
    const int resy = blockIdx.y * blockDim.y + threadIdx.y;
    if (resx > res.width || resy > res.height)
        return;
    const int x = resx;
    const int y = resy + source.botPad;
    const int ch = blockIdx.z;
    if (ch != 3)
    {
        float sum = 0.0;
        const int kernelSize = kernel.size;
        for (int cx = -kernelSize / 2; cx < kernelSize / 2 + 1; ++cx) {
            for (int cy = -kernelSize / 2; cy < kernelSize / 2 + 1; ++cy) {
                if (x + cx >= 0 && x + cx < source.width && y + cy >= 0 && y + cy < source.height) {
                    sum += kernel(cx, cy) * source(x + cx, y + cy, ch);
                }
            }
        }
        res(resx, resy, ch) = sum;
    }
    else
    {
        res(resx, resy, ch) = source(x, y, ch);
    }
}

__global__
void convTex(hipTextureObject_t tex, unsigned firstLine, unsigned imgWidth,
    unsigned imgHeight, unsigned imgCh, Image<byte> res, const Kernel kernel)
{
    const int resx = blockIdx.x * blockDim.x + threadIdx.x;
    const int resy = blockIdx.y * blockDim.y + threadIdx.y;
    if (resx > res.width || resy > res.height)
        return;
    const int x = resx;
    const int y = resy + firstLine;
    const int ch = blockIdx.z;
    if (ch != 3)
    {
        float sum = 0.0;

        const int kernelSize = kernel.size;
        for (int cx = -kernelSize / 2; cx < kernelSize / 2 + 1; ++cx) {
            for (int cy = -kernelSize / 2; cy < kernelSize / 2 + 1; ++cy) {
                /*if (x + cx >= 0 && x + cx < imgWidth && y + cy >= 0 && y + cy < imgHeight)*/ {
                    byte imgVal = tex3D<byte>(tex, x + cx, y + cy, ch);
                    sum += kernel(cx, cy) * imgVal;
                }
            }
        }
        res(resx, resy, ch) = sum;
    }
    else
    {
        res(resx, resy, ch) = tex3D<byte>(tex, x, y, ch);
    }
}

__global__
void convShKernel(const Image<byte> source, Image<byte> res, const Kernel kernel)
{
    __shared__ float kernelSh[32 * 32];
    const int resx = blockIdx.x * blockDim.x + threadIdx.x;
    const int resy = blockIdx.y * blockDim.y + threadIdx.y;
    if (resx > res.width || resy > res.height)
        return;
    const int x = resx;
    const int y = resy + source.botPad;
    const int ch = blockIdx.z;

    if (threadIdx.x < kernel.size && threadIdx.y < kernel.size && ch == 0)
    {
        const int idx = threadIdx.y * kernel.size + threadIdx.x;
        kernelSh[idx] = kernel.data[idx];
    }
    __syncthreads();
    if (ch != 3)
    {
        float sum = 0.0;

        const int kernelSize = kernel.size;
        for (int cx = -kernelSize / 2; cx < kernelSize / 2 + 1; ++cx) {
            for (int cy = -kernelSize / 2; cy < kernelSize / 2 + 1; ++cy) {
                if (x + cx >= 0 && x + cx < source.width && y + cy >= 0 && y + cy < source.height) {
                    const int kernelIdx = (cy + kernel.size / 2) * kernel.size + cx + kernel.size / 2;
                    sum += kernelSh[kernelIdx] * source(x + cx, y + cy, ch);
                }
            }
        }
        res(resx, resy, ch) = sum;
    }
    else
    {
        res(resx, resy, ch) = source(x, y, ch);
    }
}

__global__ 
void convSh(const Image<byte> source, Image<byte> res, const Kernel kernel)
{
    const int kernelSize = kernel.size;
    const int kernelRadius = (kernelSize - 1) / 2;
    const int x0 = blockIdx.x * blockDim.x;
    const int y0 = blockIdx.y * blockDim.y;
    const int resx = blockIdx.x * blockDim.x + threadIdx.x;
    const int resy = blockIdx.y * blockDim.y + threadIdx.y;
    if (resx > res.width || resy > res.height)
        return;
    const int ch = blockIdx.z;
    const int tileWidth = blockDim.x + kernelSize - 1;
    const int tileHeight = blockDim.y + kernelSize - 1;
    const int tileSize = tileWidth * tileHeight;
    const int imgWidth = source.width;
    const int imgHeight = source.height;

    extern __shared__ byte blockTile[];
    const int blockSize = blockDim.x * blockDim.y;
    const int count = tileSize / blockSize;
    const int threadNum = threadIdx.y * blockDim.x + threadIdx.x;
    int curTileIdx;
    for (int i = 0; i < count; ++i)
    {
        curTileIdx = threadNum * count + i;
        const int imgX = x0 - kernelRadius + curTileIdx % tileWidth;
        const int imgY = y0 - kernelRadius + curTileIdx / tileWidth + source.botPad;
        if (imgX < 0 || imgX >= source.width || imgY < 0 || imgY >= source.height)
            blockTile[curTileIdx] = 0;
        else
            blockTile[curTileIdx] = source(imgX, imgY, ch);
    }
    curTileIdx = count * blockSize + threadNum;
    if (tileSize > curTileIdx)
    {
        const int imgX = x0 - kernelRadius + curTileIdx % tileWidth;
        const int imgY = y0 - kernelRadius + curTileIdx / tileWidth;
        if (imgX < 0 || imgX >= imgWidth || imgY < 0 || imgY >= imgHeight)
            blockTile[count * blockSize + threadNum] = 0;
        else
            blockTile[count * blockSize + threadNum] = source(imgX, imgY, ch);
    }
    const int invId = blockSize - threadNum - 1;
    float* kernelSh = (float*)(blockTile + tileSize * sizeof(byte));
    if (invId < kernelSize * kernelSize)
    {
        kernelSh[invId] = kernel.data[invId];
    }
    __syncthreads();
    if (ch != 3)
    {
        float sum = 0.0;

        for (int cx = -kernelRadius; cx < kernelRadius + 1; ++cx) {
            for (int cy = -kernelRadius; cy < kernelRadius + 1; ++cy) {
                {
                    int tileIdx = (kernelRadius + cy + threadIdx.y) * tileWidth +
                        cx + threadIdx.x + kernelRadius;
                    int kernelIdx = (cy + kernelRadius) * kernelSize + cx + kernelRadius;
                    sum += kernelSh[kernelIdx] * blockTile[tileIdx];
                }
            }
        }
        res(resx, resy, ch) = sum;
    }
    else
    {
        int tileIdx = (kernelRadius + threadIdx.y) * tileWidth +
            threadIdx.x + kernelRadius;
        res(resx, resy, ch) = blockTile[tileIdx];
    }
}

int deviceCount;

void handleConvolutionSize(int size);



int main()
{
    hipGetDeviceCount(&deviceCount);
    deviceCount = 1;
    handleConvolutionSize(3);
    handleConvolutionSize(5);
    handleConvolutionSize(7);
    handleConvolutionSize(11);
    handleConvolutionSize(13);
    handleConvolutionSize(15);
    handleConvolutionSize(17);
    handleConvolutionSize(19);
    handleConvolutionSize(21);
    handleConvolutionSize(23);
    handleConvolutionSize(25);
    handleConvolutionSize(27);
    handleConvolutionSize(29);
    handleConvolutionSize(31);
    writeToFile("..\\results.txt");
}

std::pair<Image<byte>::Images, Image<byte>::Images> splitImages(const Image<byte>& src, const Image<byte>& dest, int devCount)
{
    return std::pair<Image<byte>::Images, Image<byte>::Images>();
}

hipTextureObject_t create3DTextureObject(const Image<byte>& img) {

    // Размеры 3D текстуры (ширина, высота, число каналов)
    const int width = img.width,
        height = img.height,
        channels = img.channels;
    hipExtent extent = make_hipExtent(width, height, channels);

    // Описание формата канала
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<byte>();

    // Создание 3D массива
    hipArray_t cuArray3D;
    hipMalloc3DArray(&cuArray3D, &channelDesc, extent);

    // Преобразование данных в формат, совместимый с 3D массивом
    std::vector<byte> linearData(img.getSize());
    for (int z = 0; z < channels; ++z) {
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                int idx = (y * width + x) * channels + z;
                int linearIdx = z * (width * height) + y * width + x;
                linearData[linearIdx] = img.data[idx];
            }
        }
    }

    // Копирование данных в 3D массив
    hipMemcpy3DParms copyParams = {};
    copyParams.srcPtr = make_hipPitchedPtr(
        linearData.data(), width * sizeof(unsigned char), width, height);
    copyParams.dstArray = cuArray3D;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    // Настройка ресурса текстуры
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray3D;

    // Настройка параметров текстуры
    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeClamp; // Адресация по ширине
    texDesc.addressMode[1] = hipAddressModeClamp; // Адресация по высоте
    texDesc.addressMode[2] = hipAddressModeClamp; // Адресация по глубине (каналы)
    texDesc.filterMode = hipFilterModePoint;     // Без интерполяции
    texDesc.readMode = hipReadModeElementType;    // Чтение байтов без нормализации
    texDesc.normalizedCoords = false;             // Не нормализовать координаты

    // Создание текстурного объекта
    hipTextureObject_t texObject = 0;
    hipCreateTextureObject(&texObject, &resDesc, &texDesc, nullptr);

    return texObject;
}

void handleConvolutionSize(int size)
{
    
    ReadWriteImg reader;
    const std::string base = "..\\";  // Получаем базовую директорию
    const std::string img_input_name = "test.png";
    const std::string img_output_name = "res.png";

    std::string full_path_to_image = base + img_input_name;
    std::string full_path_to_result = base + img_output_name;

    // Используем полный путь к изображению
    auto _img = reader.readImage(full_path_to_image);

    auto _resImg = _img.createSimilar();
    Kernel ker = KernelGenerator().generateBlurKernel(size);
    //Kernel ker = KernelGenerator().generateEdgeKernel();
    auto images = _img.splitWithOverlap(deviceCount, ker.size / 2);
    auto resImages = _resImg.splitImage(deviceCount);

    std::vector<byte*> imgDs, resDs;
    std::vector<float*> kernelDs;
    std::vector<dim3> blockDims, gridDims;
    std::vector<unsigned> memSizes;
    std::vector<DimParams> dimParamsS;
    std::vector<hipTextureObject_t> texObjs;

    for (int i = 0; i < deviceCount; ++i)
    {
        imgDs.emplace_back();
        resDs.emplace_back();
        kernelDs.emplace_back();
        blockDims.emplace_back();
        gridDims.emplace_back();
        memSizes.emplace_back();
        dimParamsS.emplace_back();
        texObjs.emplace_back();
        INITIALIZE_ENTITIES(i);
        hipSetDevice(i);


        hipMalloc(&imgD, img.getSize());
        hipMemcpy(imgD, img.data, img.getSize(), hipMemcpyHostToDevice);

        hipMalloc(&resD, img.getSize());

        unsigned size2 = ker.size * ker.size * sizeof(float);
        hipMalloc(&kernelD, size2);
        auto res = hipMemcpy(kernelD, ker.data, size2, hipMemcpyHostToDevice);

        unsigned numOfThreadsPerDim = 32;

        texObj = create3DTextureObject(img);

        blockDim.x = std::min(numOfThreadsPerDim, img.width);
        blockDim.y = std::min(numOfThreadsPerDim, img.height);
        gridDim.x = (img.width + blockDim.x - 1) / blockDim.x;
        gridDim.y = (img.height + blockDim.y - 1) / blockDim.y;
        gridDim.z = img.channels;

        memSize = (numOfThreadsPerDim + ker.size - 1) *
            (numOfThreadsPerDim + ker.size - 1) * sizeof(byte) +
            size2;

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        if (memSize > deviceProp.sharedMemPerBlock)
        {
            printf("Not enough shared memory\n");
            return;
        }

        dimParams.blockDim = blockDim;
        dimParams.gridDim = gridDim;
    }
    std::vector<std::thread> workers;
    std::vector<std::mutex> mutexes(deviceCount);
    for (int i = 0; i < deviceCount; ++i)
    {
        workers.emplace_back([&, i]() {
            auto& img = images[i];
            auto& resImg = resImages[i];
            auto& imgD = imgDs[i];
            auto& resD = resDs[i];
            auto& kernelD = kernelDs[i];
            auto& blockDim = blockDims[i];
            auto& gridDim = gridDims[i];
            auto& memSize = memSizes[i];
            auto& dimParams = dimParamsS[i];
            auto& texObj = texObjs[i];
            hipSetDevice(i);
            std::unique_lock<std::mutex> lk(mutexes[0]);
            //test common memory convolution
            Image<byte> sourceD = img;
            sourceD.data = imgD;
            auto resImgD = resImg;
            resImgD.data = resD;
            auto kerD = ker;
            kerD.data = kernelD;
            float elapsedTimeCommonMemory1 = testKernel(conv, dimParams, sourceD, resImgD, kerD);
            printf("Elapsed time for common memory %d: %f\n", i, elapsedTimeCommonMemory1);

            //test texture memory convolution
            float elapsedTimeCommonMemory2 = testKernel(convTex, dimParams, texObj, img.botPad, img.width,
                img.height, img.channels, resImgD, kerD);
            printf("Elapsed time for texture memory %d: %f\n", i, elapsedTimeCommonMemory2);

            ////test shared memory convolution
            float elapsedTimeCommonMemory3 = testKernel(convShKernel, dimParams, sourceD, resImgD, kerD);
            printf("Elapsed time for shared kernel %d: %f\n", i, elapsedTimeCommonMemory3);

            ////test shared memory convolution
            dimParams.sharedMemSize = memSize;
            float elapsedTimeCommonMemory4 = testKernel(convSh, dimParams, sourceD, resImgD, kerD);
            printf("Elapsed time for shared memory %d: %f\n", i, elapsedTimeCommonMemory4);



            {
                std::unique_lock<std::mutex> lk(results[i].mtx);
                results[i].meas[0][kerD.size] = elapsedTimeCommonMemory1;
                results[i].meas[1][kerD.size] = elapsedTimeCommonMemory2;
                results[i].meas[2][kerD.size] = elapsedTimeCommonMemory3;
                results[i].meas[3][kerD.size] = elapsedTimeCommonMemory4;
            }
            hipMemcpy(resImg.data, resD, resImg.getSize(), hipMemcpyDeviceToHost);

            hipFree(imgD);
            hipFree(resD);
            hipFree(kernelD);
            hipDestroyTextureObject(texObj);
            });
    }

    for (int i = 0; i < deviceCount; ++i)
        workers[i].join();
    //auto resImg = _img;
    //reader.writeImage(_resImg, (base + img_output_name));
    _resImg.clear();
    _img.clear();
    free(ker.data);
    return;
}

void writeToFile(const std::string& filename) {
    std::ofstream outFile(filename);
    if (!outFile) {
        std::cerr << "Ошибка открытия файла для записи!" << std::endl;
        return;
    }

    for (size_t i = 0; i < sizeof(results) / sizeof(Measurements); ++i) {
        Measurements& currResults = results[i];
        std::lock_guard<std::mutex> lock(currResults.mtx);

        for (size_t j = 0; j < 4; ++j) {
            const Measurements::MeasRes& measMap = currResults.meas[j];
            for (const auto& entry : measMap) {
                outFile << i << "," << j << "," << entry.first << "," << entry.second << std::endl;
            }
        }
    }

    outFile.close();
    std::cout << "Данные успешно записаны в файл '" << filename << "'" << std::endl;
}